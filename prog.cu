#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define MAX(x, y) ((x) > (y) ? (x) : (y))

// CUDA Kernel for LCS computation
__global__ void compute_lcs(char *X, char *Y, int *C, int m, int n, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = k - i;

    if (i <= m && j >= 0 && j <= n) {
        if (i == 0 || j == 0) {
            C[i * (n + 1) + j] = 0;
        } else if (X[i - 1] == Y[j - 1]) {
            C[i * (n + 1) + j] = C[(i - 1) * (n + 1) + (j - 1)] + 1;
        } else {
            C[i * (n + 1) + j] = MAX(C[(i - 1) * (n + 1) + j], C[i * (n + 1) + (j - 1)]);
        }
    }
}

// Function to read a string from a file
char *read_string_from_file(const char *filename, int *length) {
    FILE *file = fopen(filename, "r");
    if (!file) {
        perror("Error opening file");
        exit(EXIT_FAILURE);
    }

    fseek(file, 0, SEEK_END);
    *length = ftell(file);
    fseek(file, 0, SEEK_SET);

    char *str = (char *)malloc((*length + 1) * sizeof(char));
    fread(str, 1, *length, file);
    str[*length] = '\0';
    fclose(file);
    return str;
}

int main() {
    int m, n;
    char *X, *Y;
    char filenameX[256], filenameY[256];

    printf("Enter filename for string X: ");
    scanf("%s", filenameX);
    printf("Enter filename for string Y: ");
    scanf("%s", filenameY);

    X = read_string_from_file(filenameX, &m);
    Y = read_string_from_file(filenameY, &n);

    // Allocate memory on host
    int *C = (int *)malloc((m + 1) * (n + 1) * sizeof(int));

    // Allocate memory on device
    char *d_X, *d_Y;
    int *d_C;
    hipMalloc((void **)&d_X, m * sizeof(char));
    hipMalloc((void **)&d_Y, n * sizeof(char));
    hipMalloc((void **)&d_C, (m + 1) * (n + 1) * sizeof(int));

    // Copy data to device
    hipMemcpy(d_X, X, m * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y, n * sizeof(char), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, (m + 1) * (n + 1) * sizeof(int));

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((m + threadsPerBlock.x - 1) / threadsPerBlock.x);

    // Measure execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int k = 0; k <= m + n; ++k) {
        compute_lcs<<<blocksPerGrid, threadsPerBlock>>>(d_X, d_Y, d_C, m, n, k);
        hipDeviceSynchronize();
    }

    // Measure stop time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);

    printf("Execution Time: %f ms\n", time);

    // Copy result back to host
    hipMemcpy(C, d_C, (m + 1) * (n + 1) * sizeof(int), hipMemcpyDeviceToHost);

    printf("Length of LCS: %d\n", C[m * (n + 1) + n]);

    // Free memory
    free(X);
    free(Y);
    free(C);
    hipFree(d_X);
    hipFree(d_Y);
    hipFree(d_C);

    return 0;
}